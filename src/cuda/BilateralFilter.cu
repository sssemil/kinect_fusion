#include "hip/hip_runtime.h"
#include <BilateralFilter.cuh>

__device__ float bilateral(const float* depthmap, int width, int height, float x, float y) {
    // Perform bilinear interpolation
    int x0 = static_cast<int>(x);
    int y0 = static_cast<int>(y);
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    if (x0 < 0 || x1 >= width || y0 < 0 || y1 >= height)
        return 0.0f; // Outside the depth map, return default value

    float dx = x - x0;
    float dy = y - y0;

    float q11 = depthmap[y0 * width + x0];
    float q21 = depthmap[y0 * width + x1];
    float q12 = depthmap[y1 * width + x0];
    float q22 = depthmap[y1 * width + x1];

    float result = q11 * (1 - dx) * (1 - dy) + q21 * dx * (1 - dy) + q12 * (1 - dx) * dy + q22 * dx * dy;

    return result;
}

__global__ void applyBilateralKernel(const float* depthmap, int width, int height, float* output) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        output[y * width + x] = bilateral(depthmap, width, height, x, y);
    }
}

void applyBilateral(const std::vector<float>&depthmap, int width, int height, std::vector<float>&output) {
    float *dev_depthmap, *dev_output;
    hipMalloc((void **)&dev_depthmap, width * height * sizeof(float));
    hipMalloc((void **)&dev_output, width * height * sizeof(float));

    hipMemcpy(dev_depthmap, depthmap.data(), width * height * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    applyBilateralKernel<<<gridSize, blockSize>>>(dev_depthmap, width, height, dev_output);

    hipMemcpy(output.data(), dev_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_depthmap);
    hipFree(dev_output);
}

