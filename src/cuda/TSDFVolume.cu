#include "hip/hip_runtime.h"
#define __HIPCC__ 1
#define __host__
#define __device__
#define __global__
#define __forceinline__
#define __shared__
inline void __syncthreads() {}
using blockDim = struct { int x; int y; };
using threadIdx = struct { int x; int y; int z; };
using blockIdx = struct { int x; int y; int z; };

#include <PointCloud.h>


__global__ void integrate(const float* points,
                          const float* normals,
                          size_t pointCloudSize,
                          float truncationDistance,
                          Voxel* voxels,
                          int width, int height, int depth,
                          float voxelSize) {
     int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < pointCloudSize) {
        const float* point = &points[index * 3];
        const float* normal = &normals[index * 3];

        // Transform point to TSDF grid coordinates
        int voxelCoordX = static_cast<int>((point[0] + 2.0f) / 4.0f * width);
        int voxelCoordY = static_cast<int>((point[1] + 2.0f) / 4.0f * height);
        int voxelCoordZ = static_cast<int>((point[2] + 2.0f) / 4.0f * depth);

        // Update voxel if within TSDF volume bounds
        if (voxelCoordX >= 0 && voxelCoordX < width &&
            voxelCoordY >= 0 && voxelCoordY < height &&
            voxelCoordZ >= 0 && voxelCoordZ < depth) {
            int voxelIndex = voxelCoordX + voxelCoordY * width + voxelCoordZ * width * height;
            Voxel& voxel = voxels[voxelIndex];

            // Compute signed distance and update voxel
            float sdf = normal[0] * (point[0] - voxelCoordX * voxelSize) +
                        normal[1] * (point[1] - voxelCoordY * voxelSize) +
                        normal[2] * (point[2] - voxelCoordZ * voxelSize);
            sdf = fminf(fmaxf(sdf, -truncationDistance), truncationDistance);

            // Weighted average update
            float wNew = 1.0f;  // Example: constant weight
            voxel.distance = (voxel.distance * voxel.weight + sdf * wNew) /
                             (voxel.weight + wNew);
            voxel.weight += wNew;
        }
    }
}