#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <Eigen/Dense> // Assuming you're using Eigen for Vector3f and Matrix4f
#include <iostream>
#include <cmath>
#include <PointCloud.h>
#include <TSDFVolume.h>
// Define Vector3f and Matrix4f types if not already defined
using Vector3f = Eigen::Vector3f;
using Matrix4f = Eigen::Matrix4f;
using Vector3i = Eigen::Vector3i;

__global__ void transformPointsKernel(const Vector3f* sourcePoints, const Matrix4f pose, Vector3f* transformedPoints, int numPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        const auto rotation = pose.block(0, 0, 3, 3);
        const auto translation = pose.block(0, 3, 3, 1);
        transformedPoints[idx] = rotation * sourcePoints[idx] + translation;
    }
}

std::vector<Vector3f> transformPoints(const std::vector<Vector3f>& sourcePoints, const Matrix4f& pose) {
    int numPoints = sourcePoints.size();
    std::vector<Vector3f> transformedPoints(numPoints);

    // Allocate memory on the GPU
    Vector3f* d_sourcePoints;
    Vector3f* d_transformedPoints;
    hipMalloc((void**)&d_sourcePoints, numPoints * sizeof(Vector3f));
    hipMalloc((void**)&d_transformedPoints, numPoints * sizeof(Vector3f));

    // Copy sourcePoints from host to device
    hipMemcpy(d_sourcePoints, sourcePoints.data(), numPoints * sizeof(Vector3f), hipMemcpyHostToDevice);

    // Calculate block and grid dimensions
    int blockSize = 256;
    int numBlocks = (numPoints + blockSize - 1) / blockSize;

    // Launch kernel
    transformPointsKernel<<<numBlocks, blockSize>>>(d_sourcePoints, pose, d_transformedPoints, numPoints);

    // Copy result from device to host
    hipMemcpy(transformedPoints.data(), d_transformedPoints, numPoints * sizeof(Vector3f), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_sourcePoints);
    hipFree(d_transformedPoints);

    return transformedPoints;
}



__global__ void transformNormalsKernel(const Vector3f* sourceNormals, const Matrix4f pose, Vector3f* transformedNormals, int numNormals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numNormals) {
        const auto rotation = pose.block(0, 0, 3, 3);
        transformedNormals[idx] = rotation.inverse().transpose() * sourceNormals[idx];
    }
}

std::vector<Vector3f> transformNormals(const std::vector<Vector3f>& sourceNormals, const Matrix4f& pose) {
    int numNormals = sourceNormals.size();
    std::vector<Vector3f> transformedNormals(numNormals);

    // Allocate memory on the GPU
    Vector3f* d_sourceNormals;
    Vector3f* d_transformedNormals;
    hipMalloc((void**)&d_sourceNormals, numNormals * sizeof(Vector3f));
    hipMalloc((void**)&d_transformedNormals, numNormals * sizeof(Vector3f));

    // Copy sourceNormals from host to device
    hipMemcpy(d_sourceNormals, sourceNormals.data(), numNormals * sizeof(Vector3f), hipMemcpyHostToDevice);

    // Calculate block and grid dimensions
    int blockSize = 256;
    int numBlocks = (numNormals + blockSize - 1) / blockSize;

    // Launch kernel
    transformNormalsKernel<<<numBlocks, blockSize>>>(d_sourceNormals, pose, d_transformedNormals, numNormals);

    // Copy result from device to host
    hipMemcpy(transformedNormals.data(), d_transformedNormals, numNormals * sizeof(Vector3f), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_sourceNormals);
    hipFree(d_transformedNormals);

    return transformedNormals;
}




struct Match {
    int idx;
    float weight;
    // Add other members if needed
};

__global__ void pruneCorrespondencesKernel(const Vector3f* sourceNormals, const Vector3f* targetNormals, Match* matches, unsigned nPoints) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nPoints) {
        Match& match = matches[idx];
        if (match.idx >= 0) {
            const auto& sourceNormal = sourceNormals[idx];
            const auto& targetNormal = targetNormals[match.idx];

            // Calculate the angle between normals in degrees
            float angle = acosf(sourceNormal.dot(targetNormal) / (sourceNormal.norm() * targetNormal.norm())) * (180.0f / M_PI);

            // Invalidate the match if the angle is greater than 60 degrees
            if (angle > 60.0f) {
                match.idx = -1;
            }
        }
    }
}

void pruneCorrespondences(const std::vector<Vector3f>& sourceNormals, const std::vector<Vector3f>& targetNormals, std::vector<Match>& matches) {
    const unsigned nPoints = sourceNormals.size();

    // Allocate memory on the GPU
    Vector3f* d_sourceNormals;
    Vector3f* d_targetNormals;
    Match* d_matches;
    hipMalloc((void**)&d_sourceNormals, nPoints * sizeof(Vector3f));
    hipMalloc((void**)&d_targetNormals, targetNormals.size() * sizeof(Vector3f));
    hipMalloc((void**)&d_matches, matches.size() * sizeof(Match));

    // Copy sourceNormals, targetNormals, and matches from host to device
    hipMemcpy(d_sourceNormals, sourceNormals.data(), nPoints * sizeof(Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(d_targetNormals, targetNormals.data(), targetNormals.size() * sizeof(Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(d_matches, matches.data(), matches.size() * sizeof(Match), hipMemcpyHostToDevice);

    // Calculate block and grid dimensions
    int blockSize = 256;
    int numBlocks = (nPoints + blockSize - 1) / blockSize;

    // Launch kernel
    pruneCorrespondencesKernel<<<numBlocks, blockSize>>>(d_sourceNormals, d_targetNormals, d_matches, nPoints);

    // Copy result from device to host
    hipMemcpy(matches.data(), d_matches, matches.size() * sizeof(Match), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_sourceNormals);
    hipFree(d_targetNormals);
    hipFree(d_matches);
}



// __global__ void integrateKernel(const Vector3f* points, const Vector3f* normals, const Eigen::Matrix4f pose, const int width, const int height, const int depth, const float truncationDistance, TSDFVolume::Voxel* voxels) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < points.size()) {
//         Eigen::Vector3f point = pose * points[idx];
//         Eigen::Vector3f normal = pose.rotation() * normals[idx];

//         // Transform point to TSDF grid coordinates
//         Vector3i voxelCoord = getVoxelCoordinatesForWorldCoordinates(point);

//         // Update voxel if within TSDF volume bounds
//         if (voxelCoord[0] >= 0 && voxelCoord[0] < width && voxelCoord[1] >= 0 &&
//             voxelCoord[1] < height && voxelCoord[2] >= 0 &&
//             voxelCoord[2] < depth) {
//             int index = toLinearIndex(voxelCoord[0], voxelCoord[1], voxelCoord[2]);
//             TSDFVolume::Voxel& voxel = voxels[index];

//             // Compute signed distance and update voxel
//             float sdf = normal.dot(point);
//             sdf = min(max(sdf, -truncationDistance), truncationDistance);

//             // Weighted average update
//             float wNew = 1.0f;  // Example: constant weight
//             atomicAdd(&(voxel.distance), (voxel.distance * voxel.weight + sdf * wNew) / (voxel.weight + wNew));
//             atomicAdd(&(voxel.weight), wNew);
//         }
//     }
// }

// void integrate(const PointCloud& pointCloud, const Eigen::Matrix4f& pose, float truncationDistance) {
//     // Camera transformation
//     const Eigen::Affine3f transform(pose);

//     // Iterate over each point in the PointCloud
//     const auto& points = pointCloud.getPoints();
//     const auto& normals = pointCloud.getNormals();

//     // Allocate memory on the GPU
//     Vector3f* d_points;
//     Vector3f* d_normals;
//     hipMalloc((void**)&d_points, points.size() * sizeof(Vector3f));
//     hipMalloc((void**)&d_normals, normals.size() * sizeof(Vector3f));
//     hipMemcpy(d_points, points.data(), points.size() * sizeof(Vector3f), hipMemcpyHostToDevice);
//     hipMemcpy(d_normals, normals.data(), normals.size() * sizeof(Vector3f), hipMemcpyHostToDevice);

//     TSDFVolume::Voxel* d_voxels;
//     hipMalloc((void**)&d_voxels, width * height * depth * sizeof(TSDFVolume::Voxel));
//     hipMemcpy(d_voxels, voxels, width * height * depth * sizeof(TSDFVolume::Voxel), hipMemcpyHostToDevice);

//     // Calculate block and grid dimensions
//     int blockSize = 256;
//     int numBlocks = (points.size() + blockSize - 1) / blockSize;

//     // Launch kernel
//     integrateKernel<<<numBlocks, blockSize>>>(d_points, d_normals, pose, width, height, depth, truncationDistance, d_voxels);

//     // Copy result from device to host
//     hipMemcpy(voxels, d_voxels, width * height * depth * sizeof(TSDFVolume::Voxel), hipMemcpyDeviceToHost);

//     // Free device memory
//     hipFree(d_points);
//     hipFree(d_normals);
//     hipFree(d_voxels);
// }